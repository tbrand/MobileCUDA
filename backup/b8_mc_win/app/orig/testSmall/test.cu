#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

/**
   1.5[MB]
   div == 4, size = * 48000

   2.0[MB]
   div == 8, size = * 32000

   2.4[MB]
   div == 8, size = * 37000
**/

__global__ void __add(float* a,float* b,int size,int div){

  int idx = blockDim.x * blockIdx.x + threadIdx.x;

#pragma unroll
  for(int i = 0 ; i < div ; i ++){
    a[idx + (size*i)/div] += b[idx + (size*i)/div];
  }

}

static float elapsed(struct timeval tv0,struct timeval tv1){
  return (float)(tv1.tv_sec - tv0.tv_sec)
    + (float)(tv1.tv_usec - tv0.tv_usec)
    * 0.000001f;
}

int main(){

  struct timeval t0,t1;

  gettimeofday(&t0,NULL);

  float *h_a = NULL;
  float *h_b = NULL;
  float *d_a = NULL;
  float *d_b = NULL;

  int div = 4;
  int threadNum = 1024;

  unsigned int size = (threadNum*div) * 48000;

  int blockNum  = size/(threadNum*div);

  printf("blockNum  : %d\n",blockNum);
  printf("threadNum : %d\n",threadNum);
  printf("size      : %d\n",size);

  printf("vector size : %d\n",sizeof(float)*size);

  int ite = 6000;

  hipMalloc((void**)&d_a,sizeof(float)*size);
  hipMalloc((void**)&d_b,sizeof(float)*size);

  h_a = (float*)malloc(sizeof(float)*size);
  h_b = (float*)malloc(sizeof(float)*size);

  for(int i = 0 ; i < size ; i ++){
    h_a[i] = 0.0f;
    h_b[i] = 1.0f;
  }

  dim3 threads(threadNum,1,1);
  dim3 blocks(blockNum,1,1);

  hipMemcpy(d_a,h_a,sizeof(float)*size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,sizeof(float)*size,hipMemcpyHostToDevice);

  for(int i = 0 ; i < ite ; i ++){
    __add<<<blocks,threads>>>(d_a,d_b,size,div);
  }

  hipMemcpy(h_a,d_a,sizeof(float)*size,hipMemcpyDeviceToHost);

  int pass = 1;
  int firstFailedIndex = 0;

  for(int i = 0 ; i < size ; i ++){

    //    printf("h_a[%d]:%f ",i,h_a[i]);

    if(h_a[i] != ite){
      firstFailedIndex = i;
      pass = 0;
      break;
    }
  }

  if(pass){
    printf("Result test PASS!\n");
  }else{
    printf("Result test Failed\n");
    printf("h_a[%d] == %f\n",firstFailedIndex,h_a[firstFailedIndex]);
  }

  gettimeofday(&t1,NULL);

  printf("TIME RESULT : %f\n",elapsed(t0,t1));

  return 0;

}
