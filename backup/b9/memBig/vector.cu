/**
   Sample for Mobile CUDA
   Simple Adding Vectors Application.

   Authoer @ Taichirou Suzuki
**/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

/**
   Simple Kernel.
**/
__global__ void ___add(float* a,float* b,unsigned long size){
  int _x = blockDim.x * blockIdx.x + threadIdx.x;
  int _y = blockDim.y * blockIdx.y + threadIdx.y;
  
  unsigned long id = _x + _y * size;

  a[id] += b[id];
}

static float elapsed(struct timeval tv0,struct timeval tv1){
  return (float)(tv1.tv_sec - tv0.tv_sec)
    + (float)(tv1.tv_usec - tv0.tv_usec)
    * 0.000001f;
}

int main(void){

  struct timeval t0,t1;

  gettimeofday(&t0,NULL);
  
  /**
     Define Vector Size.
  **/
  //  unsigned long _hen = 11000;
  unsigned long _hen = 18000;
  //  unsigned long _hen = 18000;
  unsigned long size = _hen * _hen;
  printf("gyouretu size : %lu\n",size);
  /**
     Number Of Launch Kernel.
  **/
  int numOfLaunchKernel = 1;
  //int numOfLaunchKernel = 1;

  hipSetDevice(0);

  //  float* h_a = (float*)malloc(sizeof(float)*size);
  //  float* h_b = (float*)malloc(sizeof(float)*size);

  float* d_a = NULL;
  float* d_b = NULL;
  //  float* d_c = NULL;

  hipMalloc((void**)&d_a,sizeof(float)*size);
  hipMalloc((void**)&d_b,sizeof(float)*size);
  //  cudaMalloc((void**)&d_c,sizeof(float)*size);

  float* h_a = NULL;
  float* h_b = NULL;

  /*
  cudaError_t res;
  res = cudaHostAlloc((void **)&h_a,sizeof(float)*size,0);
  printf("cudaHostAlloc : %d\n",res);
  res = cudaHostAlloc((void **)&h_b,sizeof(float)*size,0);
  printf("cudaHostAlloc : %d\n",res);
  */

  h_a = (float*)malloc(sizeof(float)*size);
  h_b = (float*)malloc(sizeof(float)*size);  

  //  float* h_c = (float*)malloc(sizeof(float)*size);

  printf("This Sample Application Uses %d[Mbyte] per vector.(Total : %d[Mbyte])\n",sizeof(float)*size >> 20,sizeof(float)*size*2 >> 20);

  for(int i =  0 ; i < size ; i ++){
    h_a[i] = 0.0f;
    h_b[i] = 1.0f;
  }

  //  int ite = 140;
  int ite = 155;
  //  int ite = 1000000;

  for(int j = 0 ; j < ite ; j ++){

    hipMemcpy(d_a,h_a,sizeof(float)*size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,sizeof(float)*size,hipMemcpyHostToDevice);

    int _size = 10;
    dim3 threads(_size,_size,1);
    dim3 grid(_hen/_size,_hen/_size,1);

    for(int i = 0 ; i < numOfLaunchKernel ; i ++){
      //__add<<<grid,threads>>>(d_c,d_a,d_b,_hen);
      ___add<<<grid,threads>>>(d_a,d_b,_hen);
      /**
	 Main thread can sleep at here.
      **/
      //    sleep(1);
    }

    //  cudaMemcpy(h_c,d_c,sizeof(float)*size,cudaMemcpyDeviceToHost);
    hipMemcpy(h_a,d_a,sizeof(float)*size,hipMemcpyDeviceToHost);

  }

  int pass = 1;
  for(int i = 0 ; i < size ; i ++){
    //    if(h_c[i] != numOfLaunchKernel){
    //    if(h_a[i] != numOfLaunchKernel){
    //    printf("H_A[%d] : %d",i,h_a[i]);
    if(h_a[i] != ite){
      pass = 0;
    }
  }

  if(pass){
    printf(">Result TEST : PASS\n");
  }else{
    printf(">Result TEST : FAILED\n");
  }

  hipFree(d_a);
  hipFree(d_b);
  //  cudaFree(d_c);

  free(h_a);
  free(h_b);
  //  cudaFreeHost(h_a);
  //  cudaFreeHost(h_b);
  //  free(h_c);

  printf("Application Closed...\n");

  gettimeofday(&t1,NULL);

  printf("My RESULT : %f\n",elapsed(t0,t1));

  return 0;
}
