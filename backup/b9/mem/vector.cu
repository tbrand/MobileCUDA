/**
   Sample for Mobile CUDA
   Simple Adding Vectors Application.

   Authoer @ Taichirou Suzuki
**/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>


__global__ void ___add(float* a,float* b,unsigned long size){

  unsigned int _x = blockDim.x * blockIdx.x + threadIdx.x;

  a[_x] += b[_x];
}

static float elapsed(struct timeval tv0,struct timeval tv1){
  return (float)(tv1.tv_sec - tv0.tv_sec)
    + (float)(tv1.tv_usec - tv0.tv_usec)
    * 0.000001f;
}

int main(void){

  struct timeval t0,t1;

  gettimeofday(&t0,NULL);

  unsigned long _hen = 16000;
  //  unsigned long size = _hen * _hen;
  unsigned long size = 16000*16000;

  printf("gyouretu size : %lu\n",size);

  int numOfLaunchKernel = 1;

  hipSetDevice(0);

  float* d_a = NULL;
  float* d_b = NULL;
  //  float* d_c = NULL;

  hipMalloc((void**)&d_a,sizeof(float)*size);
  hipMalloc((void**)&d_b,sizeof(float)*size);

  float* h_a = NULL;
  float* h_b = NULL;

  hipError_t res;
  res = hipHostAlloc((void **)&h_a,sizeof(float)*size,0);
  printf("hipHostAlloc : %d\n",res);
  res = hipHostAlloc((void **)&h_b,sizeof(float)*size,0);
  printf("hipHostAlloc : %d\n",res);

  printf("This Sample Application Uses %d[Mbyte] per vector.(Total : %d[Mbyte])\n",sizeof(float)*size >> 20,sizeof(float)*size*2 >> 20);

  for(int i =  0 ; i < size ; i ++){
    h_a[i] = 0.0f;
    h_b[i] = 1.0f;
  }

  int ite = 190;

  int _size = 10;
  //  dim3 threads(_size,_size,1);
  //  dim3 grid(_hen/_size,_hen/_size,1);
  dim3 threads(10,1,1);
  dim3 grid(size,1,1);

  printf("SIZE  :  %d\n",size);
  
  for(int j = 0 ; j < ite ; j ++){

    hipMemcpy(d_a,h_a,sizeof(float)*size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,sizeof(float)*size,hipMemcpyHostToDevice);

    //    ___add<<<grid,threads>>>(d_a,d_b,_hen);
    ___add<<<grid,threads>>>(d_a,d_b,1);

    hipMemcpy(h_a,d_a,sizeof(float)*size,hipMemcpyDeviceToHost);

    printf("Add[%d]\n",j);

    printf("\tH_A[0] : %d\n",h_a[0]);

    sleep(5);
  }


  int pass = 1;
  for(int i = 0 ; i < size ; i ++){
    //    printf("H_A[%d] : %d\n",i,h_a[i]);
  }

  if(pass){
    printf(">Result TEST : PASS\n");
  }else{
    printf(">Result TEST : FAILED\n");
  }

  hipFree(d_a);
  hipFree(d_b);
  //  cudaFree(d_c);

  free(h_a);
  free(h_b);
  //  cudaFreeHost(h_a);
  //  cudaFreeHost(h_b);
  //  free(h_c);

  printf("Application Closed...\n");

  gettimeofday(&t1,NULL);

  printf("My RESULT : %f\n",elapsed(t0,t1));

  return 0;
}
