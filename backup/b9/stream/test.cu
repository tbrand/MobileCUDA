#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

__global__ void __add(float* a,float* b,int size,int blockNum,int streamNum,int streamNo){

  int streamLength = size/streamNum;
  int blockSize    = streamLength/blockNum;
  int threadSize   = blockSize/blockDim.x;

  int startIndex   = streamLength*streamNo + blockSize*blockIdx.x + threadSize*threadIdx.x;

  for(int i = 0 ; i < threadSize ; i ++){
    a[startIndex+i] += b[startIndex+i];
  }
}

static float elapsed(struct timeval tv0,struct timeval tv1){
  return (float)(tv1.tv_sec - tv0.tv_sec)
    + (float)(tv1.tv_usec - tv0.tv_usec)
    * 0.000001f;
}

int main(){

  struct timeval t0,t1;

  gettimeofday(&t0,NULL);

  float *h_a = NULL;
  float *h_b = NULL;
  float *d_a = NULL;
  float *d_b = NULL;

  int streamNum = 8;
  int threadNum = 1024;
  int blockNum  = 1;

  int size = streamNum * threadNum * blockNum * 40000;

  int ite = 90;

  hipMalloc((void**)&d_a,sizeof(float)*size);
  hipMalloc((void**)&d_b,sizeof(float)*size);

  h_a = (float*)malloc(sizeof(float)*size);
  h_b = (float*)malloc(sizeof(float)*size);

  for(int i = 0 ; i < size ; i ++){
    h_a[i] = 0.0f;
    h_b[i] = 1.0f;
  }

  dim3 threads(threadNum,1,1);
  dim3 blocks(blockNum,1,1);

  hipMemcpy(d_a,h_a,sizeof(float)*size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,sizeof(float)*size,hipMemcpyHostToDevice);

  printf("size == %d\n",size);

  hipStream_t* streams;

  streams = (hipStream_t*)malloc(sizeof(hipStream_t)*streamNum);

  for(int i = 0 ; i < streamNum ; i ++){

    hipStreamCreate(&streams[i]);

  }

  for(int i = 0 ; i < ite ; i ++){
    for(int j = 0 ; j < streamNum ; j ++){
      __add<<<blocks,threads,0,streams[j]>>>(d_a,d_b,size,blockNum,streamNum,j);
    }
  }

  for(int i = 0 ; i < streamNum ; i ++){

    hipStreamSynchronize(streams[i]);
    hipStreamDestroy(streams[i]);

  }

  hipMemcpy(h_a,d_a,sizeof(float)*size,hipMemcpyDeviceToHost);

  int pass = 1;
  int firstFailedIndex = 0;

  for(int i = 0 ; i < size ; i ++){

    if(h_a[i] != ite){
      firstFailedIndex = i;
      pass = 0;
    }
  }

  if(pass){
    printf("Result test PASS!\n");
  }else{
    printf("Result test Failed\n");
    printf("h_a[%d] == %f\n",firstFailedIndex,h_a[firstFailedIndex]);
  }

  gettimeofday(&t1,NULL);

  printf("TIME RESULT : %f\n",elapsed(t0,t1));

  return 0;

}
