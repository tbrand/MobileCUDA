#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

#define VSIZE 1024*50000
#define TSIZE 1024
#define BSIZE VSIZE/TSIZE
#define ITE 10

__global__ void add(float* a,float* b){

  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  b[idx] += 1;
  a[idx] += b[idx];

}

int main(){

  float *ha,*hb;
  float *da,*db;

  ha = (float*)malloc(sizeof(float)*VSIZE);
  hb = (float*)malloc(sizeof(float)*VSIZE);

  hipMalloc((void**)&da,sizeof(float)*VSIZE);
  hipMalloc((void**)&db,sizeof(float)*VSIZE);

  for(int i = 0 ; i < VSIZE ; i ++){
    ha[i] = 0.0f;
    hb[i] = 0.0f;
  }

  hipMemcpy(da,ha,sizeof(float)*VSIZE,hipMemcpyHostToDevice);
  hipMemcpy(db,hb,sizeof(float)*VSIZE,hipMemcpyHostToDevice);

  dim3 threads(TSIZE,1,1);
  dim3 blocks (BSIZE,1,1);

  printf("threads : %d\n",threads.x);
  printf("blocks  : %d\n",blocks.x);

  for(int i = 0 ; i < ITE ; i ++){
    add<<<blocks,threads>>>(da,db);
    hipDeviceSynchronize();
  }

  hipMemcpy(ha,da,sizeof(float)*VSIZE,hipMemcpyDeviceToHost);

  for(int i = 0 ; i < VSIZE ; i ++){
    if(ha[i] != ((ITE+1)*ITE)/2 ){
      printf("ha[%d]\t%f\n",i,ha[i]);
      printf("Result TEST : FAILED\n");
      exit(-1);
    }
  }

  printf("Result TEST : PASS\n");

  free(ha);
  free(hb);
  hipFree(da);
  hipFree(db);

  return 0;
}
