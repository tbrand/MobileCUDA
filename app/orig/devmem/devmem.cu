#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

#define THREADS 1024
#define SIZE    (THREADS*250000)

__device__ float device_array0[SIZE];
__device__ float device_array1[SIZE];

__global__ void devmem_kernel(){

  int id = blockDim.x*blockIdx.x + threadIdx.x;

  device_array0[id] += 1.0f;
  device_array1[id] += device_array0[id];

}

static float elapsed(struct timeval tv0,struct timeval tv1){
  return (float)(tv1.tv_sec - tv0.tv_sec)
    + (float)(tv1.tv_usec - tv0.tv_usec)
    * 0.000001f;
}

int main(){

  struct timeval t0,t1;

  gettimeofday(&t0,NULL);

  printf("Vector SIZE : %d[Mbyte]\n",sizeof(float)*SIZE >> 20);


  float *result;
  int   niter;
  hipError_t res
;
  result = (float*)malloc(sizeof(float)*SIZE);

  dim3  threads (THREADS,1,1);
  dim3  blocks  (SIZE/THREADS,1,1);

  niter = 2000;

  for(int i = 0 ; i < SIZE ; i ++){
    result[i] = 0.0f;
  }

  res = hipMemcpyToSymbol(HIP_SYMBOL(device_array0),result,sizeof(float)*SIZE,0,hipMemcpyHostToDevice);
  res = hipMemcpyToSymbol(HIP_SYMBOL(device_array1),result,sizeof(float)*SIZE,0,hipMemcpyHostToDevice);

  printf("cudaMemcpyToSymbol(%d)\n",res);

  for(int i = 0 ; i < niter ; i ++){

    devmem_kernel<<<blocks,threads>>>();

    hipDeviceSynchronize();

  }

  res = hipMemcpyFromSymbol(result,HIP_SYMBOL(device_array1),sizeof(float)*SIZE,0,hipMemcpyDeviceToHost);

  printf("cudaMemcpyFromSymbol(%d)\n",res);

  int pass = 1;

  for(int i = 0 ; i < SIZE ; i ++){
    if(result[i] != (float)(((niter+1)*niter)/2.0f) ){
      pass = 0;
      printf("result[%d] : %f\n",i,result[i]);
      break;
    }
  }

  if(pass){
    printf("Result test : PASS\n");
  }else{
    printf("Result test : Failed\n");
  }

  gettimeofday(&t1,NULL);

  printf("TIME RESULT : %f[sec](DEV MEM)\n",elapsed(t0,t1));

  return 0;
}
