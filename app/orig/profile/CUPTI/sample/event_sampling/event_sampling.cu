#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2013 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain profiler
 * event values by sampling.
 */


#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include <stdio.h> 
#include <hip/hip_runtime_api.h>
#include <cupti_events.h>

#ifdef _WIN32
#include <windows.h>
#else
#include <unistd.h>
#include <pthread.h>
#endif

#define CHECK_CU_ERROR(err, cufunc)                                     \
  if (err != hipSuccess)                                              \
    {                                                                   \
      printf ("Error %d for CUDA Driver API function '%s'.\n",          \
              err, cufunc);                                             \
      exit(-1);                                                         \
    }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                       \
  if (err != CUPTI_SUCCESS)                                     \
    {                                                           \
      const char *errstr;                                       \
      cuptiGetResultString(err, &errstr);                       \
      printf ("%s:%d:Error %s for CUPTI API function '%s'.\n",  \
              __FILE__, __LINE__, errstr, cuptifunc);           \
      exit(-1);                                                 \
    }

#define EVENT_NAME "inst_executed"
#define N 100000
#define TESLA_ITERS 10000
#define FERMI_ITERS 10000
#define SAMPLE_PERIOD_MS 50

// used to signal from the compute thread to the sampling thread
static volatile int testComplete = 0;

static hipCtx_t context;
static hipDevice_t device;
static char *eventName;


// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  for(int n = 0 ; n < 100; n++) {
    if (i < size)
      C[i] = A[i] + B[i];
  }
}

static void
initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

void *
sampling_func(void *arg)
{
  CUptiResult cuptiErr;
  CUpti_EventGroup eventGroup;
  CUpti_EventID eventId;
  size_t bytesRead; 
  uint64_t eventVal;

  cuptiErr = cuptiSetEventCollectionMode(context, 
                                         CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiSetEventCollectionMode");

  cuptiErr = cuptiEventGroupCreate(context, &eventGroup, 0);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupCreate");

  cuptiErr = cuptiEventGetIdFromName(device, eventName, &eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGetIdFromName");

  cuptiErr = cuptiEventGroupAddEvent(eventGroup, eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupAddEvent");

  cuptiErr = cuptiEventGroupEnable(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupEnable");

  while (!testComplete) {
    bytesRead = sizeof(eventVal);
    cuptiErr = cuptiEventGroupReadEvent(eventGroup, 
                                        CUPTI_EVENT_READ_FLAG_NONE, 
                                        eventId, &bytesRead, &eventVal);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupReadEvent");
    if (bytesRead != sizeof(eventVal)) {
      printf("Failed to read value for \"%s\"\n", eventName);
      exit(-1);
    }

    printf("%s: %llu\n", eventName, (unsigned long long)eventVal);

#ifdef _WIN32
    Sleep(SAMPLE_PERIOD_MS);
#else
    usleep(SAMPLE_PERIOD_MS * 1000);
#endif
  }

  cuptiErr = cuptiEventGroupDisable(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDisable");

  cuptiErr = cuptiEventGroupDestroy(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDestroy");

  return NULL;
}

static void
compute(int iters)
{
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int sum, i;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel (multiple times to make sure we have time for
  // sampling)
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  //  for (i = 0; i < iters; i++) {
  for (i = 0; i < iters; i++) {
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  }

    
  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      printf("kernel execution FAILED\n");
      exit(-1);
    }
  }
}

int
main(int argc, char *argv[])
{
#ifdef _WIN32
  HANDLE hThread;
#else
  int status;
  pthread_t pThread;
#endif
  hipError_t err;
  int computeCapabilityMajor=0;
  int computeCapabilityMinor=0;
  int deviceNum;
  int deviceCount;
  char deviceName[32];

  printf("Usage: %s [device_num] [event_name]\n", argv[0]);
    
  err = hipInit(0);
  CHECK_CU_ERROR(err, "hipInit");

  err = hipGetDeviceCount(&deviceCount);
  CHECK_CU_ERROR(err, "hipGetDeviceCount");

  if (deviceCount == 0) {
    printf("There is no device supporting CUDA.\n");
    exit(-1);
  }

  if (argc > 1)
    deviceNum = atoi(argv[1]);
  else
    deviceNum = 0;
  printf("CUDA Device Number: %d\n", deviceNum);

  err = hipDeviceGet(&device, deviceNum);
  CHECK_CU_ERROR(err, "hipDeviceGet");

  err = hipDeviceGetName(deviceName, 32, device);
  CHECK_CU_ERROR(err, "hipDeviceGetName");

  printf("CUDA Device Name: %s\n", deviceName);

  err = hipDeviceComputeCapability(&computeCapabilityMajor, 
                                  &computeCapabilityMinor, 
                                  device);
  CHECK_CU_ERROR(err, "hipDeviceComputeCapability");

  if (argc > 2) {
    eventName = argv[2];
  }
  else {
    if (computeCapabilityMajor > 1) {
      eventName = EVENT_NAME;
    } 
    else {
      printf("Event sampling is not supported for Tesla family devices.\n");
      return 0;
    }
  }

  err = hipCtxCreate(&context, 0, device);
  CHECK_CU_ERROR(err, "hipCtxCreate");

  
  testComplete = 0;

  printf("Creating sampling thread\n");
#ifdef _WIN32
  hThread = CreateThread(NULL, 0, (LPTHREAD_START_ROUTINE) sampling_func,
                         NULL, 0, NULL );
  if (!hThread) {
    printf("CreateThread failed\n");
    exit(-1);
  }
#else
  status = pthread_create(&pThread, NULL, sampling_func, NULL);
  if (status != 0) {
    perror("pthread_create");
    exit(-1);
  }
#endif

  // run kernel while sampling
  compute((computeCapabilityMajor > 1) ? FERMI_ITERS : TESLA_ITERS);

  // "signal" the sampling thread to exit and wait for it
  testComplete = 1;
#ifdef _WIN32
  WaitForSingleObject(hThread, INFINITE);
#else
  pthread_join(pThread, NULL);
#endif

  hipDeviceSynchronize();
  return 0;
}
