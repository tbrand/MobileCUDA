#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

int main(int argc,char* argv[]){

  hipError_t res;
  float* d;
  int i,j;
  size_t pitch,width,height;

  for(i = 0 ; i < 1000 ; i ++){

    for(j = 0 ; j < 100 ; j ++){

      width = 5*i;
      height = 100*j;

      res = hipMallocPitch(&d,&pitch,width,height);

      //      size_t expected_pitch = width*height != 0 ? ((width*height-1)/(512*100)+1)*512 : 0;

      size_t expected_pitch;

      expected_pitch = width*height != 0 ? (width/512 + 1)*512 : 0;

      //      printf("pitch : %lu\n",pitch);
      //      printf("expected_pitch : %lu\n",expected_pitch);

      if(pitch != expected_pitch){
	printf("width : %lu\n",width);
	printf("height : %lu\n",height);
	exit(1);
      }

      if(res != hipSuccess){
	printf("What ... ?(%d)\n",res);
	exit(1);
      }

      if(expected_pitch != pitch){

	printf("pitch == %d\n",pitch);
	printf("expected pitch == %d\n",expected_pitch);
	printf("Total device memory usage : %d\n",width*height);

      }
    }
    
    hipFree(d);
    
  }

  return 0;
}
