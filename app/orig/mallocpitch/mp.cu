#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define THREADS 1024

__global__ void kernel(float* d_1,float* d_2){

  int id;

  id = blockDim.x*blockIdx.x + threadIdx.x;

  d_1[id] += 1.0f;
  d_2[id] += d_1[id];
}

int main(){

  hipError_t res;
  float *d_1,*d_2,*h;
  size_t pitch;
  size_t size = THREADS * 10;
  int i,ite = 1000;

  h = (float*)malloc(sizeof(float)*size*size);

  for(i = 0 ; i < size*size ; i ++)
    h[i] = 0.0f;

  res = hipMallocPitch(&d_1,&pitch,sizeof(float)*size,size);
  
  if(res != hipSuccess){
    printf("Oops ...\n");
    exit(-1);
  }

  res = hipMallocPitch(&d_2,&pitch,sizeof(float)*size,size);
  
  if(res != hipSuccess){
    printf("Oops ...\n");
    exit(-1);
  }

  printf("pitch : %lu\n",pitch);
  printf("height : %lu\n",size);
  printf("region : %lu[MB]\n",pitch*size >> 20);

  res = hipMemcpy2D(d_1,pitch,h,sizeof(float)*size,sizeof(float)*size,size,hipMemcpyHostToDevice);

  if(res != hipSuccess){
    printf("Oops ...\n");
    exit(-1);
  }

  res = hipMemcpy2D(d_2,pitch,h,sizeof(float)*size,sizeof(float)*size,size,hipMemcpyHostToDevice);

  if(res != hipSuccess){
    printf("Oops ...\n");
    exit(-1);
  }

  dim3 threads(THREADS,1,1);
  dim3 blocks(size/THREADS,1,1);

  for(i = 0 ; i < ite ; i ++){

    kernel<<<blocks,threads>>>(d_1,d_2);

  }

  res = hipMemcpy(h,d_2,sizeof(float)*size*size,hipMemcpyDeviceToHost);

  if(res != hipSuccess){
    printf("Oops ...\n");
    exit(-1);
  }

  for(i = 0 ; i < size ; i ++){
    if(h[i] != ((ite+1)*ite)/2.0f  ){
      printf("h[%d] == %f\n",i,h[i]);
      exit(-1);
    }
  }

  sleep(10);

  return 0;
}
