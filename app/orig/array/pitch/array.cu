#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <nvml.h>

#define IS 5000
#define JS 5000
#define I 50000
#define J 50000

int main(){

  /*Create context*/
  float* d;
  hipMalloc((void**)&d,sizeof(float)*1);
  hipFree(d);

  FILE *fp;
  char *fname = "array_data.csv";

  fp = fopen(fname,"w");

  int failed_counter = 0;

  hipError_t res;
  hipArray_t array;

  nvmlReturn_t nres;
  nvmlMemory_t mem;
  nvmlDevice_t dev;
  
  size_t width = 0,height = 0;
  size_t prev_width = 0,prev_height = 0;
  struct hipChannelFormatDesc desc;

  size_t before_mem = 0;
  size_t after_mem  = 0;
  size_t prev_mem   = 0;

  desc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
  //  desc = cudaCreateChannelDesc(16,16,16,16,cudaChannelFormatKindSigned);

  nres = nvmlInit();

  if(nres != NVML_SUCCESS){
    printf("NVML ERROR : %d\n",nres);
    exit(-1);
  }

  nres = nvmlDeviceGetHandleByIndex(0,&dev);

  if(nres != NVML_SUCCESS){
    printf("NVML ERROR : %d\n",nres);
    exit(-1);
  }
  
#if 1

  int max = 0 , min = 100000000;

  //  for(int i = IS ; i < IS+I ; i ++){
  for(int i = IS ; i < IS+I ; i += 32){

    width = i;

    //    for(int j = JS ; j < JS+J ; j ++){
    for(int j = JS ; j < JS+J ; j += 32){

      height = j;
      
      nres = nvmlDeviceGetMemoryInfo(dev,&mem);

      if(nres != NVML_SUCCESS){
	printf("NVML ERROR : %d\n",nres);
	exit(-1);
      }

      before_mem = mem.free;

      res = hipMallocArray(&array,&desc,width,height,0);

      if(res == hipSuccess){

	nres = nvmlDeviceGetMemoryInfo(dev,&mem);
	
	if(nres != NVML_SUCCESS){
	  printf("NVML ERROR : %d\n",nres);
	  exit(-1);
	}

	after_mem = mem.free;

	size_t used;
	size_t expected;

	used = before_mem-after_mem;

	expected = ((desc.w+desc.x+desc.y+desc.z+7)/8)
	  *((width+31)&~(size_t)31)
	  *((height+127)&~(size_t)127)
	  + (2<<20);

	if(expected < used){
	  printf("expected : %lu\n",expected);
	  printf("used     : %lu\n",used);
	  printf("width    : %lu\n",width);
	  printf("height   : %lu\n",height);
	  exit(-1);
	}

	if(min > expected-used){
	  min = expected-used;
	}

	if(max < expected-used){
	  max = expected-used;
	}

	printf("%lu\t%lu\t%lu\t%lu\t%lu\n",expected-used,min,max,width,height);

	/*
	fprintf(fp,"%lu",used);

	if(j < J-1){
	  fprintf(fp,",");
	}else{
	  fprintf(fp,"\n");
	}
	*/

	/*

	size_t expected;

	if(before_mem-after_mem != expected){
	  printf("Failed\n");
	  printf("width        : %lu\n",width);
	  printf("height       : %lu\n",height);
	  printf("width*height : %lu\n",width*height);
	  printf("used         : %lu\n",before_mem-after_mem);
	  printf("be expected as : %lu\n",expected);
	  if(++failed_counter >= 10)
	    exit(-1);
	}else{
	  //	  printf("PASS(%lu,%lu:%lu)",width,height,expected);
	}

	size_t e;

	e = ((width-1)/32+1)*131072+((height-1)/128+1)*524288;
	
	expected = e > 2097152 ? e : 2097152;

	if(expected-used > max){
	  max = expected-used;
	}

	if(min > (int)expected-(int)used){
	  min = expected-used;
	}

	printf("max:%d\tmin:%d\tused:%lu\texpected:%lu\twidth:%d\theight:%d\n",max,min,used,expected,width,height);

	if(min < 0){
	  printf("Detected min < 0\n");
	  printf("width:%d,height:%d,used:%d,expected:%d\n",width,height,used,expected);
	  exit(-1);
	}

	//	size_t left  = used/w-used/2048;
	//	size_t right = -960+1966080/w;

	if(left != right){
	  printf("\t%lu\t%lu\n",left,right);
	  exit(-1);
	}

	*/

	if((before_mem-after_mem) > prev_mem){

	  /*
	  printf("width  : %lu\n",width);
	  printf("height : %lu\n",height);
	  printf("used   : %lu\n",before_mem-after_mem);
	  printf("diff(width)  : %lu\n",width-prev_width);
	  printf("diff(height) : %lu\n",height-prev_height);
	  printf("diff         : %lu\n",(before_mem-after_mem)-prev_mem);
	  printf("diff(width*height) : %lu\n",width*height-prev_width*prev_height);
	  printf("(used-2097152)/131072: %f\n",(float)(before_mem-after_mem-2097152)/(float)131072);
	  */

	  prev_mem = before_mem-after_mem;
	  prev_width = width;
	  prev_height = height;
	}
	

	hipFreeArray(array);

      }else{

	printf("\nERROR : %d\n",res);
	printf("Failed Counter : %d\n",failed_counter);
	exit(-1);
	
      }
    }
  }

#endif

  return 0;
}
