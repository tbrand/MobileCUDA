#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

__global__ void kernel(){
  
}

int main(){

  float *d;
  hipError_t res;

  res = hipMalloc((void**)&d,sizeof(float)*100);
  
  printf("Address : %p(res: %d)\n",d,res);

  res = hipDeviceReset();

  printf("device reset : %d\n",res);

  /*malloc array*/
  hipArray_t array;
  const struct hipChannelFormatDesc desc;
  size_t width,height;

  width = 100;
  height = 100;

  desc.w = 10;
  desc.x = 10;
  desc.y = 10;
  desc.z = 10;

  res = hipMallocArray(&array,&desc,width,height,0);

  printf("hipMallocArray : %d\n",res);
  printf("x: %d, y: %d, z: %d, w: %d\n",desc.x,desc.y,desc.z,desc.w);

  res = hipMalloc((void**)&d,sizeof(float)*100);

  printf("Address : %p(res: %d)\n",d,res);  
  
  return 0;
}
