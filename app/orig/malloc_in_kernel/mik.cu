#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

__global__ void malloc_in_kernel(float* d_a,int n,int length){

  float* x;

  length = 5;

  x = (float*)malloc(sizeof(float)*length);

  for(int i = 0 ; i < length ; i ++){
    x[i] = 1.0f;
  }

  int length_per_block;
  int length_per_thread;
  int start,end;

  length_per_block  = n/gridDim.x;
  length_per_thread = length_per_block/blockDim.x;

  start = length_per_block*blockIdx.x + length_per_thread*threadIdx.x;
  end   = length_per_block*blockIdx.x + length_per_thread*(threadIdx.x+1);

  for(int i = start ; i < end ; i ++){
    for(int j = 0 ; j < length ; j ++){
      d_a[i] += x[j];
    }
  }
  
  free(x);
}

static float elapsed(struct timeval tv0,struct timeval tv1){
  return (float)(tv1.tv_sec - tv0.tv_sec)
    + (float)(tv1.tv_usec - tv0.tv_usec)
    * 0.000001f;
}

int main(){

  struct timeval t0,t1;

  gettimeofday(&t0,NULL);

  int n = 600000000;
  int iter = 2;
  int length = 5;

  int thread_num = 8;
  int block_num  = 4;
  
  dim3 threads(thread_num,1,1);
  dim3 blocks(block_num,1,1);

  float* h_a;
  float* d_a;

  h_a = (float*)malloc(sizeof(float)*n);

  hipMalloc((void**)&d_a,sizeof(float)*n);
  hipMemcpy(d_a,h_a,sizeof(float)*n,hipMemcpyHostToDevice);

  for(int i = 0 ; i < iter ; i ++){
    h_a[i] = 0.0f;
  }

  for(int i = 0 ; i < iter ; i ++){

    malloc_in_kernel<<<blocks,threads>>>(d_a,n,length);

  }

  hipMemcpy(h_a,d_a,sizeof(float)*n,hipMemcpyDeviceToHost);

  int test = 1;

  for(int i = 0 ; i < n ; i ++){
    if(h_a[i] != iter*length){
      test = 0;
      break;
    }
  }

  if(test){
    printf("Result test PASS\n");
  }else{
    printf("Result test Failed\n");
  }

  gettimeofday(&t1,NULL);

  printf("TIME RESULT : %f[sec](MIK)\n",elapsed(t0,t1));
}
