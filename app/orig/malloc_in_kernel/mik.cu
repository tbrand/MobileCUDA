#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

__global__ void malloc_in_kernel(int n){

  float* x;

  x = (float*)malloc(sizeof(float)*n);

  for(int i = 0 ; i < n ; i ++){
    x[i] = 1.0f;
  }
}

int main(){

  int n = 1000000;
  int iter = 1000;
  
  dim3 threads(1024,1,1);
  dim3 blocks(100,1,1);

  for(int i = 0 ; i < iter ; i ++){

    malloc_in_kernel<<<blocks,threads>>>(n);

  }

  sleep(10);

  printf("FIN\n");

}
