#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define THREADS 1024

__global__ void kernel(struct hipPitchedPtr pitchedPointer){

  int id;
  float *d;

  id = blockDim.x*blockIdx.x + threadIdx.x;

  d = (float*)pitchedPointer.ptr;

  d[id] += 1.0f;

}

int main(){

  int i,ite = 100,size = 40;

  hipError_t res;
  struct hipExtent extent;
  struct hipPitchedPtr pitchedPointer;
  float* ans;

  ans = (float*)malloc(sizeof(float)*size*size*size*THREADS);

  for(i = 0 ; i < size*size*size*THREADS ; i ++){
    ans[i] = 0.0f;
  }

  extent.width = sizeof(float)*size*THREADS;
  extent.height = size;
  extent.depth = size;

  res = hipMalloc3D(&pitchedPointer,extent);

  if(res != hipSuccess){
    printf("Oh...(%d)\n",res);
    exit(-1);
  }

  printf("Pointer : %p\n",pitchedPointer.ptr);

  res = hipMemcpy(pitchedPointer.ptr,ans,sizeof(float)*size*size*size*THREADS,hipMemcpyHostToDevice);

  if(res != hipSuccess){
    printf("Oh...(%d)\n",res);
    exit(-1);
  }

  dim3 threads(THREADS,1,1);
  dim3 blocks(size*size*size,1,1);

  for(i = 0 ; i < ite ; i ++){
    kernel<<<blocks,threads>>>(pitchedPointer);
  }

  res = hipMemcpy(ans,pitchedPointer.ptr,sizeof(float)*size*size*size*THREADS,hipMemcpyDeviceToHost);
  
  if(res != hipSuccess){
    printf("Oops ...\n");
    exit(-1);
  }

  for(i = 0 ; i < size*size*size*THREADS ; i ++){
    if(ans[i] != ite){
      printf("ans[%d] == %f\n",i,ans[i]);
      exit(-1);
    }
  }

  printf("Elements : %d\n",size*size*size*THREADS);
  sleep(10);

  res = hipFree(pitchedPointer.ptr);

  if(res != hipSuccess){
    printf("Oops ...\n");
    exit(-1);
  }

  return 0;
}
