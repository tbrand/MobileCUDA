#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

int main(int argc,char* argv[]){

  hipError_t res;
  int i,j,k;

  size_t width,height,depth,prevpitch = 512;
  size_t prevwidth=0,prevheight=0,prevdepth=0;

  struct hipPitchedPtr pitchedDevPtr;
  struct hipExtent extent;

  for(i = 0 ; i < 10000 ; i += 100){
    for(j = 0 ; j < 2000 ; j += 10){
      for(k = 0 ; k < 2000 ; k += 10){

	width = 1*(i+1);
	height = 1*(j+1);
	depth = 1*(k+1);

	extent.width = width;
	extent.height = height;
	extent.depth = depth;

	res = hipMalloc3D(&pitchedDevPtr,extent);

	if(res != hipSuccess){

	  printf("extent.width  : %lu\n",extent.width);
	  printf("extent.height : %lu\n",extent.height);
	  printf("extent.depth  : %lu\n",extent.depth);
	  
	  printf("What ...?(%d)\n",res);
	  exit(-1);

	}else{

	  if(prevpitch != pitchedDevPtr.pitch){
	    printf("comfirm differing pitch\n");
	    printf("width:%lu\theight:%lu\tdepth:%lu\tpitch%lu\n",prevwidth,prevheight,prevdepth,pitchedDevPtr.pitch);
	    printf("width:%lu\theight:%lu\tdepth:%lu\tpitch%lu\n",width,height,depth,pitchedDevPtr.pitch);
	    prevpitch = pitchedDevPtr.pitch;
	  }
	}

	prevwidth = width;
	prevheight = height;
	prevdepth = depth;

	hipFree(pitchedDevPtr.ptr);

	size_t expected_pitch;

	expected_pitch = (width/512 + 1)*512;

	if(expected_pitch != pitchedDevPtr.pitch){
	  printf("Oh ...\n");
	  exit(-1);
	}

	/*
	size_t expected_pitch;

	//    expected_pitch = extent.width*extent.height*extent.depth != 0 ? ((extent.width*extent.height*extent.depth-1)/(5120000)+1)*512 : 0;

	expected_pitch = extent.width*extent.height != 0 ? ((extent.width*extent.height-1)/(51200*extent.depth)+1)*512 : 0;

	if(expected_pitch != pitchedDevPtr.pitch){

	  //      printf("extent.width  : %lu\n",extent.width);
	  //      printf("extent.height : %lu\n",extent.height);
	  //      printf("extent.depth  : %lu\n",extent.depth);
	  printf("width*height*depth : %lu\n",extent.width*extent.height*extent.depth);

	  //    printf("\txsize : %lu\n",pitchedDevPtr.xsize);
	  //    printf("\tysize : %lu\n",pitchedDevPtr.ysize);
	  printf("\tpitch : %lu\n",pitchedDevPtr.pitch);
	  //      printf("\tptr   : %p \n",pitchedDevPtr.ptr);
	  //    printf("expected_pitch : %lu\n",expected_pitch);
	  //      exit(1);
	}else{
	  printf("OK *** i,j,k : %d,%d,%d\n",i,j,k);
	}
	*/
      }
    }
  }

  return 0;
}
