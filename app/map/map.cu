#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

__global__ void kernel(float* d,int size){
  
  int length_per_block;
  int length_per_thread;
  int start,end;

  length_per_block  = size/gridDim.x;
  length_per_thread = length_per_block/blockDim.x;

  start = length_per_block*blockIdx.x + length_per_thread*threadIdx.x;
  end   = length_per_block*blockIdx.x + length_per_thread*(threadIdx.x+1);

  for(int i = start ; i < end ; i ++ ){
    d[i] += 1.0f;
  }

}

int main(){

  int res,length,ite;

  ite = 10;

  res = hipSetDeviceFlags(hipDeviceMapHost);

  printf("cudaSetDeviceFlags(%d)\n",res);

  float* h;
  float* d;

  length = 1000000000;

  res = hipHostAlloc((void**)&h,sizeof(float)*length,hipHostMallocMapped|hipHostMallocPortable);

  printf("cudaHostAlloc(%d)\n",res);

  res = hipHostGetDevicePointer((void**)&d,h,0);

  printf("cudaHostGetDevicePointer(%d)\n",res);
  printf("device address : %p\n",d);

  h = (float*)malloc(sizeof(float)*length);

  for(int i = 0 ; i < length ; i ++){
    h[i] = 0.0f;
  }

  dim3 threads(1000,1,1);
  dim3 blocks(1000,1,1);

  res = hipMemcpy(d,h,sizeof(float)*length,hipMemcpyHostToDevice);

  printf("cudaMemcpyHostToDevice(%d)\n",res);

  for(int i = 0 ; i < ite ; i ++){

    kernel<<<blocks,threads>>>(d,length);

  }

  res = hipMemcpy(h,d,sizeof(float)*length,hipMemcpyDeviceToHost);

  printf("cudaMemcpyDeviceToHost(%d)\n",res);
  
  for(int i = 0 ; i < length ; i ++){
    if(h[i] != ite){
      printf("Result test : Failed\n");
      printf("h[%d] == %f\n",i,h[i]);
      return -1;
    }
  }

  printf("Result test PASS\n");
      
  return 0;

}
