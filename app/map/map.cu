#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

__global__ void kernel(){
  
}

int main(){

  int res,length;

  res = hipSetDeviceFlags(hipDeviceMapHost);

  printf("cudaSetDeviceFlags(%d)\n",res);

  float* h;
  float* d;

  length = 100;

  res = hipHostAlloc((void**)&h,sizeof(float)*length,hipHostMallocMapped|hipHostMallocPortable);

  printf("cudaHostAlloc(%d)\n",res);

  res = hipHostGetDevicePointer((void**)&d,h,0);

  printf("cudaHostGetDevicePointer(%d)\n",res);
  printf("device address : %p\n",d);

  

  return 0;

}
