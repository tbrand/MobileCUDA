#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>

__global__ void kernel(float* d,float* d1,int size){
  
  int length_per_block;
  int length_per_thread;
  int start,end;

  length_per_block  = size/gridDim.x;
  length_per_thread = length_per_block/blockDim.x;

  start = length_per_block*blockIdx.x + length_per_thread*threadIdx.x;
  end   = length_per_block*blockIdx.x + length_per_thread*(threadIdx.x+1);

  for(int i = start ; i < end ; i ++ ){
    d1[i] += 1.0f;
    d[i]  += d1[i];
  }
}

static float elapsed(struct timeval tv0,struct timeval tv1){
  return (float)(tv1.tv_sec - tv0.tv_sec)
    + (float)(tv1.tv_usec - tv0.tv_usec)
    * 0.000001f;
}

int main(){

  struct timeval t0,t1;

  gettimeofday(&t0,NULL);

  int res,length,ite;

  ite = 10;

  res = hipSetDeviceFlags(hipDeviceMapHost);

  printf("cudaSetDeviceFlags(%d)\n",res);

  float *h;
  float *d,*d1;

  length = 500000000;

  //  res = cudaHostAlloc((void**)&h,sizeof(float)*length,cudaHostAllocMapped|cudaHostAllocPortable);

  h = (float*)valloc(sizeof(float)*length);

  res = hipHostRegister(h,sizeof(float)*length,hipHostRegisterMapped);

  printf("cudaHostAlloc(%d)\n",res);

  res = hipHostGetDevicePointer((void**)&d,h,0);

  printf("cudaHostGetDevicePointer(%d)\n",res);
  printf("device address : %p\n",d);

  h = (float*)malloc(sizeof(float)*length);

  for(int i = 0 ; i < length ; i ++){
    h[i] = 0.0f;
  }

  dim3 threads(1000,1,1);
  dim3 blocks(1000,1,1);

  res = hipMemcpy(d,h,sizeof(float)*length,hipMemcpyHostToDevice);

  printf("cudaMemcpyHostToDevice(%d)\n",res);

  res = hipMalloc((void**)&d1,sizeof(float)*length);

  printf("cudaMalloc(%d) : Address %p\n",res,d1);

  res = hipMemcpy(d1,h,sizeof(float)*length,hipMemcpyHostToDevice);

  printf("cudaMemcpyHostToDevice(%d)\n",res);

  for(int i = 0 ; i < ite ; i ++){

    kernel<<<blocks,threads>>>(d,d1,length);

  }

  res = hipMemcpy(h,d,sizeof(float)*length,hipMemcpyDeviceToHost);

  printf("cudaMemcpyDeviceToHost(%d)\n",res);
  
  for(int i = 0 ; i < length ; i ++){
    if(h[i] != ((ite+1)*ite)/2 ){
      printf("Result test : Failed\n");
      printf("h[%d] == %f\n",i,h[i]);
      return -1;
    }
  }

  printf("Result test PASS\n");

  gettimeofday(&t1,NULL);

  printf("TIME RESULT : %f[sec](MAP)\n",elapsed(t0,t1));
      
  return 0;

}
